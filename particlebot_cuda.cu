#include "hip/hip_runtime.h"
#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>

#include <helper_functions.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particlebot_kernel_impl.cuh"

extern "C"
{

  void cudaInit(int argc, char **argv)
  {
    int devID;

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    devID = findCudaDevice(argc, (const char **)argv);

    if (devID < 0)
      {
        printf("No CUDA Capable devices found, exiting...\n");
        exit(EXIT_SUCCESS);
      }
  }

  void cudaGLInit(int argc, char **argv)
  {
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    findCudaGLDevice(argc, (const char **)argv);
  }

  void allocateArray(void **devPtr, size_t size)
  {
    checkCudaErrors(hipMalloc(devPtr, size));
  }

  void freeArray(void *devPtr)
  {
    checkCudaErrors(hipFree(devPtr));
  }

  void threadSync()
  {
    checkCudaErrors(hipDeviceSynchronize());
  }

  void copyArrayToDevice(void *device, const void *host, int offset, int size)
  {
    checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
  }

  void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
  {
    checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo,
                                                 cudaGraphicsMapFlagsNone));
  }

  void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
  {
    checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
  }

  void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
  {
    void *ptr;
    checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,
                                                         *cuda_vbo_resource));
    return ptr;
  }

  void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
  {
    checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
  }

  void copyArrayFromDevice(void *host, const void *device,
                           struct hipGraphicsResource **cuda_vbo_resource, int size)
  {
    if (cuda_vbo_resource)
      {
        device = mapGLBufferObject(cuda_vbo_resource);
      }

    checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

    if (cuda_vbo_resource)
      {
        unmapGLBufferObject(*cuda_vbo_resource);
      }
  }

  void setParameters(SimParams *hostParams)
  {
    // copy parameters to constant memory
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(x1obs), hostParams->x1obs, sizeof(float)*hostParams->nobstacles));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(x2obs), hostParams->x2obs, sizeof(float)*hostParams->nobstacles));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(y1obs), hostParams->y1obs, sizeof(float)*hostParams->nobstacles));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(y2obs), hostParams->y2obs, sizeof(float)*hostParams->nobstacles));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(x_cir_obs), hostParams->x_cir_obs, sizeof(float)*hostParams->n_cir_obstacles));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(y_cir_obs), hostParams->y_cir_obs, sizeof(float)*hostParams->n_cir_obstacles));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(r_cir_obs), hostParams->r_cir_obs, sizeof(float)*hostParams->n_cir_obstacles));
  }

  //Round a / b to nearest higher integer value
  uint iDivUp(uint a, uint b)
  {
    return (a % b != 0) ? (a / b + 1) : (a / b);
  }

  // compute grid and thread block size for a given number of elements
  void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
  {
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
  }

  // compute grid and thread block size for a given number of elements
  void computeGridSize2(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
  {
    numThreads = blockSize;
    numBlocks = iDivUp(n, numThreads);
  }

  void integrateSystem(float *pos,
                       float *vel,
                       float *rad,
                       float deltaTime,
                       uint nCells,
                       float time)
  {
    thrust::device_ptr<float2> d_pos2((float2 *)pos);
    thrust::device_ptr<float2> d_vel2((float2 *)vel);
    thrust::device_ptr<float> d_rad((float *)rad);

    thrust::for_each(
                     thrust::make_zip_iterator(thrust::make_tuple(d_pos2, d_vel2, d_rad)),
                     thrust::make_zip_iterator(thrust::make_tuple(d_pos2+nCells, d_vel2+nCells, d_rad+nCells)),
                     integrate_functor(deltaTime, time));
  }

  void calcHash(uint  *gridParticlebotHash,
                uint  *gridParticlebotIndex,
                float *pos,
                int    nCells)
  {
    uint numThreads, numBlocks;
    computeGridSize(nCells, 256, numBlocks, numThreads);

    // execute the kernel
    calcHashD<<< numBlocks, numThreads >>>(gridParticlebotHash,
                                           gridParticlebotIndex,
                                           (float2 *) pos
                                           );

    // check if kernel invocation generated an error
    getLastCudaError("Kernel execution failed");
  }

  void updateRad_light_wave(float *pos,
                            float *absForce_a,float *absForce_r,
                            float *rad,
                            float *phase,
                            float time, float deltaTime, int *dead,
                            int    nCells)
  {
    uint numThreads, numBlocks;
    computeGridSize(nCells, 256, numBlocks, numThreads);

    // execute the kernel
    updateRad_light_wave <<< numBlocks, numThreads >>>((float2 *)pos, (float*)absForce_a, (float*)absForce_r, (float*)rad, phase,  time, deltaTime, dead);

    // check if kernel invocation generated an error
    getLastCudaError("Kernel execution failed");
  }


  void curand_setup(hiprandState *state, int N) {
    uint numThreads, numBlocks;
    computeGridSize(N, 256, numBlocks, numThreads);
    curand_setup_kernel << < numBlocks, numThreads >> > (state, N);
  }

  void add_normal_noise(hiprandState *state, float *val, float std, int N) {
    uint numThreads, numBlocks;
    computeGridSize(N, 256, numBlocks, numThreads);
    add_normal_noise_kernel << < numBlocks, numThreads >> > (state, val, std, N);
  }

  void updatePhase(float *pos,
                   float *phase,
                   float spacing,
                   float max_d,
                   float min_d,
                   int nCells)

  {
    uint numThreads, numBlocks;
    computeGridSize(nCells, 256, numBlocks, numThreads);
    updatePhase <<< numBlocks, numThreads >>>((float2 *)pos, phase, spacing, max_d, min_d);

    // check if kernel invocation generated an error
    getLastCudaError("Kernel execution failed");
  }


  void updateCol(float *rad,
                 float *col,
                 int    nCells, float *pos, float *phase, int *dead)
  {
    uint numThreads, numBlocks;
    computeGridSize(nCells, 256, numBlocks, numThreads);

    // execute the kernel
    updateCol_k <<< numBlocks, numThreads >>>((float*)rad, (float4 *)col, (float2 *)pos, phase, dead);

    // check if kernel invocation generated an error
    getLastCudaError("Kernel execution failed");
  }

  void calcCOG(float *pos, float *temppos, float *temppos1,
               int    nCells, float time, int hist_steps, float hist_int)
  {
    int ind = ((int)(time/hist_int ) )% hist_steps;
    uint numThreads, numBlocks;
    int nCells_new;
    nCells_new = nCells;
    computeGridSize2(nCells, 64, numBlocks, numThreads);
    float mul = 1.0f/float(nCells);
    uint smemSize = sizeof(float)*2*(numThreads+1);
    // execute the kernel
    if(numBlocks==1){
      calcCOG1<64> <<< numBlocks, numThreads, smemSize >>>((float2 *)pos, (float2 *) temppos1, nCells, mul);
    }
    else{
      calcCOG<64> <<< numBlocks, numThreads, smemSize >>>((float2 *)pos, (float2 *) temppos, nCells);
    }

    while(numBlocks>1){
      nCells_new = (int)(numBlocks);

      computeGridSize2(nCells_new, 64, numBlocks, numThreads);
      smemSize = sizeof(float)*2*(numThreads+1);
      if(numBlocks== 1){
        calcCOG1<64> <<< numBlocks, numThreads, smemSize >>>((float2 *)temppos, (float2 *) temppos1, nCells_new, mul);
      }
      else{
        calcCOG<64> <<< numBlocks, numThreads, smemSize >>>((float2 *)temppos, (float2 *) temppos1, nCells_new);
        checkCudaErrors(hipMemcpy(temppos, temppos1, 2*sizeof(float)*nCells_new, hipMemcpyDeviceToDevice));
      }


    }
    checkCudaErrors(hipMemcpy(pos+2*(ind+nCells), temppos1, 2*sizeof(float), hipMemcpyDeviceToDevice));

    //float2 temp;
    //checkCudaErrors(hipMemcpy(&temp, temppos1, 2*sizeof(float), hipMemcpyDeviceToHost));
    //printf("centroid ind: %d, x: %f, y: %f\n", ind, temp.x, temp.y);

    getLastCudaError("Kernel execution failed");
  }


  void reorderDataAndFindCellStart(uint  *cellStart,
                                   uint  *cellEnd,
                                   float *sortedPos,
                                   float *sortedVel,
                                   float *sortedRad,
                                   uint  *gridParticlebotHash,
                                   uint  *gridParticlebotIndex,
                                   float *oldPos,
                                   float *oldVel,
                                   float *oldRad,
                                   uint   nCells,
                                   uint   numCells)
  {
    uint numThreads, numBlocks;
    computeGridSize(nCells, 256, numBlocks, numThreads);

    // set all cells to empty
    checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

#if USE_TEX
    checkCudaErrors(hipBindTexture(0, oldPosTex, oldPos, nCells*sizeof(float2)));
    checkCudaErrors(hipBindTexture(0, oldVelTex, oldVel, nCells*sizeof(float2)));
#endif

    uint smemSize = sizeof(uint)*(numThreads+1);
    reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
                                                                       cellStart,
                                                                       cellEnd,
                                                                       (float2 *) sortedPos,
                                                                       (float2 *) sortedVel,
                                                                       sortedRad,
                                                                       gridParticlebotHash,
                                                                       gridParticlebotIndex,
                                                                       (float2 *) oldPos,
                                                                       (float2 *) oldVel,
                                                                       oldRad
                                                                       );
    getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
    checkCudaErrors(hipUnbindTexture(oldPosTex));
    checkCudaErrors(hipUnbindTexture(oldVelTex));
#endif
  }



  void collide(float *newVel,
               float *absForce_a,
               float *absForce_r,
               float *sortedPos,
               float *sortedVel,
               float *sortedRad,
               uint  *gridParticlebotIndex,
               uint  *cellStart,
               uint  *cellEnd,
               uint   nCells,
               uint   numCells, float deltaTime)
  {
#if USE_TEX
    checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, nCells*sizeof(float4)));
    checkCudaErrors(hipBindTexture(0, oldVelTex, sortedVel, nCells*sizeof(float4)));
    checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
    checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));
#endif

    // thread per particlebot
    uint numThreads, numBlocks;
    computeGridSize(nCells, 64, numBlocks, numThreads);

    // execute the kernel
    collideD<<< numBlocks, numThreads >>>((float2 *)newVel,
                                          (float *)absForce_a,
                                          (float *)absForce_r,
                                          (float2 *)sortedPos,
                                          (float2 *)sortedVel,
                                          sortedRad,
                                          gridParticlebotIndex,
                                          cellStart,
                                          cellEnd,
                                          deltaTime);

    // check if kernel invocation generated an error
    getLastCudaError("Kernel execution failed");

#if USE_TEX
    checkCudaErrors(hipUnbindTexture(oldPosTex));
    checkCudaErrors(hipUnbindTexture(oldVelTex));
    checkCudaErrors(hipUnbindTexture(cellStartTex));
    checkCudaErrors(hipUnbindTexture(cellEndTex));
#endif
  }

  void sortParticlebots(uint *dGridParticlebotHash, uint *dGridParticlebotIndex, uint nCells)
  {
    thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticlebotHash),
                        thrust::device_ptr<uint>(dGridParticlebotHash + nCells),
                        thrust::device_ptr<uint>(dGridParticlebotIndex));
  }

}   // extern "C"
